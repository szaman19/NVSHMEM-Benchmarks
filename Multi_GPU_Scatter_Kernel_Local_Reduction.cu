#include "test_utils.hpp"
#include "NVSHMEM_Kernels.cuh"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <numeric>
#include "nvml.h"
#include "mpi.h"



int main(int argc, char* argv[]){
  int mype_node;
  hipStream_t stream;
  int rank, nranks;
  MPI_Comm mpi_comm = MPI_COMM_WORLD;
  nvshmemx_init_attr_t attr;
  
  int my_rank_in_node = std::stoi(getenv("OMPI_COMM_WORLD_LOCAL_RANK"));
  int my_device = my_rank_in_node % 4;

  CUDA_CHECK(hipSetDevice(my_device));
  
  CUDA_CHECK(hipStreamCreate(&stream));
  
  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &nranks);

  attr.mpi_comm = &mpi_comm;
  nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
  mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);

  CUDA_CHECK(hipDeviceSynchronize());
  nvshmemx_barrier_all_on_stream(stream);
  CUDA_CHECK(hipDeviceSynchronize());

  printf("[%d / %d] Cuda device set to [%d]\n", rank, nranks, my_device);
  
  Edges edges = Read_Arxiv_Data();
  constexpr int num_nodes = 169343; 

  const int input_stride = (edges.num_edges() + nranks - 1) / nranks;
  const int output_stride = (num_nodes - 1 + nranks) / nranks;

  const int input_start = input_stride * rank;
  const int output_start = output_stride * rank;

  const int num_in_rows = input_stride;
  const int num_out_rows = output_stride;

  constexpr int num_cols = 128;
  constexpr int mini_batch_size = 1;
  const auto input_matrix_size = num_cols * num_in_rows;
  const auto output_matrix_size = num_cols * num_out_rows;
  
  std::ofstream outfile;
  constexpr int count = 10;
  
  if (rank == 0){  
    std::string nranks_string = std::to_string(nranks);
    std::string output_dir = "scatter_results/";
    std::string suffix = "_scatter_arxiv_local_reduction.csv";
    std::string _fname = output_dir + nranks_string + suffix;
    outfile.open(_fname);
    for (size_t i = 0; i < count; i++){
      outfile << ", Run " << i+1;
    }
    outfile<<'\n';
  }

  std::vector<double> timing_array;

  const int num_indices = (rank == nranks-1) ? edges.num_edges() - input_start : input_stride;

  printf("[%d] Reading ARXIV indices : %d ([%d]-[%d]) \n", rank, num_indices, output_start, output_start+num_indices);

  for(size_t i = 0; i < count; ++i){

    std::cout <<"["<<rank <<"]" << " Starting generating random matrix: \t" << input_matrix_size << std::endl;
    float* value_matrix = _rand_arr_nvshmem(input_matrix_size);
    std::cout <<"["<<rank <<"]" << " Finished generating random matrix" << std::endl;
    const int* source_indices = &edges.get_source_indices()[input_start];
    float* local_indices = new float[num_indices];
    float* global_remap = new float[num_indices];

    if (rank == 0){
      std::cout << "[0] Starting remapping" << std::endl;
    }
    auto start_remap = std::chrono::steady_clock::now();
    remap_indices(source_indices, local_indices, global_remap, num_indices);
    auto elapsed_time_remap = static_cast<double>(since(start_remap).count());
    
    if (rank == 0){
      std::cout << "[0] Finished remapping" << std::endl;
    }
    float* local_indices_vec = _int2float(source_indices, num_indices);
    float* indices_vec = _device_arr(global_remap, num_indices);
    float* output_matrix = _zero_arr_nvshmem(output_matrix_size);
    float* local_value_matrix = _zero_arr_nvshmem(input_matrix_size);
    // Pad the indices_vector so    
    
    indices_vec = _pad(indices_vec, num_indices, input_stride, -1.0);
    local_indices_vec = _pad(local_indices_vec, num_indices, input_stride, -1.0);


    hipDeviceSynchronize();
    if (rank == 0){
      std::cout << "[0] Finished setup" << std::endl;
    }
    nvshmemx_barrier_all_on_stream(stream);
    if (rank == 0){
      std::cout << "[0] Starting local scatter" << std::endl;
    }
    auto start = std::chrono::steady_clock::now();
    dim3 grid, block;
    block.x = 16;
    block.y = 32;
    block.z = 1; 

    grid.x = (num_cols + block.x - 1) / block.x;
    grid.y = (num_in_rows + block.y - 1) / block.y;
    grid.z = 1; 

    scatter<<<grid, block, 0, stream>>>(value_matrix,
                                        local_indices_vec,
                                        local_value_matrix,
                                        mini_batch_size,
                                        num_in_rows,
                                        num_cols,
                                        num_out_rows);

    Scatter_NVSHMEM_Kernel<<<grid, block, 0, stream>>>(local_value_matrix,
                                                       indices_vec,
                                                       output_matrix,
                                                       mini_batch_size,
                                                       num_in_rows,
                                                       num_cols,
                                                       num_out_rows);
    
    nvshmemx_quiet_on_stream(stream);
    nvshmemx_barrier_all_on_stream(stream);
    hipStreamSynchronize(stream);

    auto elapsed_time = static_cast<double>(since(start).count());
    if (rank == 0){
      timing_array.push_back(elapsed_time + elapsed_time_remap);
      outfile<<","<<elapsed_time + elapsed_time_remap;
      std::cout << "Scatter Elapsed Time (S)=" << (elapsed_time + elapsed_time_remap) * 1E-9 << std::endl;
    }

    nvshmem_free(value_matrix);
    nvshmem_free(local_value_matrix);
    hipFree(local_indices_vec);
    hipFree(indices_vec);
    nvshmem_free(output_matrix);
    
  }

  nvshmemx_barrier_all_on_stream(stream); 
  hipDeviceSynchronize();
  if (rank == 0){
    outfile<<'\n';
    std::cout << "Average Scatter Elapsed Time (S)=" << average(timing_array) * 1E-9 << std::endl;
    outfile.close();
  }

  nvshmem_finalize();
  MPI_Finalize();
  return 0;
}

