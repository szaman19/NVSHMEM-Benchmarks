#include "test_utils.hpp"
#include "NVSHMEM_Kernels.cuh"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <numeric>
#include "nvml.h"
#include "mpi.h"



int main(int argc, char* argv[]){
  int mype_node;
  hipStream_t stream;
  int rank, nranks;
  MPI_Comm mpi_comm = MPI_COMM_WORLD;
  nvshmemx_init_attr_t attr;

  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &nranks);

  attr.mpi_comm = &mpi_comm;
  nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
  mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
  hipSetDevice(mype_node);
  hipStreamCreate(&stream);
  
  printf("[%d / %d] Cuda device set to [%d]\n", rank, nranks, mype_node);
  
  Edges edges = Read_Shift_Data();
  constexpr int num_nodes = 1166243; 
  // constexpr int num_edges = 1166243;
  const int input_stride = (edges.num_edges() + nranks - 1) / nranks;
  const int output_stride = (num_nodes - 1 + nranks) / nranks;

  const int input_start = input_stride * rank;
  const int output_start = output_stride * rank;

  const int num_in_rows = input_stride;
  const int num_out_rows = output_stride;

  constexpr int num_cols = 128;
  constexpr int mini_batch_size = 1;
  const auto input_matrix_size = num_cols * num_in_rows;
  const auto output_matrix_size = num_cols * num_out_rows;
  
  std::ofstream outfile;
  constexpr int count = 1;
  
  if (rank == 0){  
    std::string nranks_string = std::to_string(nranks);
    std::string _fname = nranks_string +"_scatter_arxiv.csv";
    outfile.open(_fname);
    for (size_t i = 0; i < count; i++){
      outfile << ", Run " << i+1;
    }
    outfile<<'\n';
  }

  std::vector<double> timing_array;

  const int num_indices = (rank == nranks-1) ? edges.num_edges() - input_start : input_stride;

  printf("[%d] Reading ARXIV indices : %d ([%d]-[%d]) \n", rank, num_indices, output_start, output_start+num_indices);

  for(size_t i = 0; i < count; ++i){
    float* value_matrix = _rand_arr_nvshmem(input_matrix_size);
    float* indices_vec = _int2float(&edges.get_source_indices()[input_start], num_indices);
    float* output_matrix = _zero_arr_nvshmem(output_matrix_size);
    
    // Pad the indices_vector so    
    
    indices_vec = _pad(indices_vec, num_indices, input_stride, -1.0);

    hipDeviceSynchronize();
//    nvshmemx_barrier_all_on_stream(stream);

    auto start = std::chrono::steady_clock::now();
    dim3 grid, block;
    block.x = 16;
    block.y = 32;
    block.z = 1; 

    grid.x = (num_cols + block.x - 1) / block.x;
    grid.y = (num_in_rows + block.y - 1) / block.y;
    grid.z = 1; 

    Scatter_NVSHMEM_Kernel<<<grid, block, 0, stream>>>(value_matrix,
                                                       indices_vec,
                                                       output_matrix,
                                                       mini_batch_size,
                                                       num_in_rows,
                                                       num_cols,
                                                       num_out_rows);
    
    // nvshmemx_quiet_on_stream(stream);
    nvshmemx_barrier_all_on_stream(stream);
    hipDeviceSynchronize();

    auto elapsed_time = static_cast<double>(since(start).count());
    if (rank == 0){
      timing_array.push_back(elapsed_time);
      outfile<<","<<elapsed_time;
      std::cout << "Scatter Elapsed Time (S)=" << elapsed_time * 1E-9 << std::endl;
    }

    nvshmem_free(value_matrix);
    hipFree(indices_vec);
    nvshmem_free(output_matrix);
    
  }

  nvshmemx_barrier_all_on_stream(stream); 
  hipDeviceSynchronize();
  if (rank == 0){
    outfile<<'\n';
    std::cout << "Average Scatter Elapsed Time (S)=" << average(timing_array) * 1E-9 << std::endl;
    outfile.close();
  }

  nvshmem_finalize();
  MPI_Finalize();
  return 0;
}

